#include "hip/hip_runtime.h"
/*  Copyright 2015 Giuseppe Bilotta, Alexis Herault, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <iostream>

#include "Bubble.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define USE_PLANES 0

#if USE_PLANES
#define USE_GHOST 1 // set to 0 for standard planes
#else
#define USE_GHOST 0 // never use ghosts unless using planes
#endif



Bubble::Bubble(GlobalData *_gdata) : XProblem(_gdata),
	dyn_layers(0)
{
	// Size and origin of the simulation domain
	R = 0.025;
	H = 10*R;
	lx = ly = 6*R;
	lz = H;

	// TODO GAUSSIAN kernel of radius 3
	SETUP_FRAMEWORK(
		formulation<SPH_GRENIER>,
		viscosity<DYNAMICVISC>,
		boundary<DYN_BOUNDARY>,
		flags<ENABLE_DTADAPT | (USE_PLANES ? ENABLE_PLANES : ENABLE_NONE)>
	);

	// SPH parameters
	// Grenier sets h/R = 0.128
	//set_deltap(6.72e-4/1.3);
	set_deltap(0.128*R/1.3);

	if (simparams()->boundarytype == DYN_BOUNDARY) {
		dyn_layers = simparams()->get_influence_layers() + 1;
		extra_offset = make_double3(dyn_layers*m_deltap);
	} else {
		dyn_layers = 0;
		extra_offset = make_double3(0.0);
	}
	m_size = make_double3(lx, ly, lz) + 2*extra_offset;
	m_origin = -m_size/2;

	simparams()->buildneibsfreq = 10;

	simparams()->tend = 1.0;

	physparams()->epsinterface = 0.08;

	// Physical parameters
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	float g = length(physparams()->gravity);

	//set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;

	physparams()->r0 = m_deltap;

	float maxvel = sqrt(g*H);
	float rho0 = 1;
	float rho1 = 1000;

	size_t air = add_fluid(rho0);
	size_t water = add_fluid(rho1);

	set_equation_of_state(air,  1.4, 198*maxvel);
	set_equation_of_state(water,  7.0f, 14*maxvel);

	set_kinematic_visc(air, 4.5e-3f);
	set_kinematic_visc(water, 3.5e-5f);

	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.01);

	// Name of problem used for directory creation
	m_name = "Bubble";

	// Building the geometry
	float r0 = physparams()->r0;

	setPositioning(PP_CORNER);
	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(m_origin),
		m_size.x,	m_size.y, m_size.z);
	disableCollisions(experiment_box);
	setMassByDensity(experiment_box, physparams()->rho0[1]);

	GeometryID fluid = addBox(GT_FLUID, FT_SOLID,
		Point(m_origin + extra_offset),
		lx, ly, H);
	// the actual particle mass will be set during the
	// initializeParticles routine
	setMassByDensity(fluid, physparams()->rho0[0]);

}

void Bubble::copy_planes(PlaneList &planes)
{
#if USE_PLANES
	// z = m_origin.z
	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	// z = m_origin.z+lz
	planes.push_back( implicit_plane(0, 0, -1.0, m_origin.z+lz) );
	// y = m_origin.y
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.y) );
	// y = m_origin.y+ly
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.y+ly) );
	// x = m_origin.x
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.x) );
	// x = m_origin.x+lx
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.x+lx) );
#endif
}


// the bubble is initially located centered at 2R from the bottom.
bool is_inside(double3 const& origin, float R, double4 const& pt)
{
	return
		(pt.x*pt.x) +
		(pt.y*pt.y) +
		(pt.z - (origin.z+2*R))*(pt.z - (origin.z+2*R)) < R*R;
}

// Mass and density initialization
	void
Bubble::initializeParticles(BufferList &buffers, const uint numParticles)
{
	// Example usage

	// 1. warn the user if this is expected to take much time
	printf("Initializing particles density and mass...\n");

	// 2. grab the particle arrays from the buffer list
	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	double4 *pos_global = buffers.getData<BUFFER_POS_GLOBAL>();
	float4 *pos = buffers.getData<BUFFER_POS>();

	// 3. iterate on the particles
	for (uint i = 0; i < numParticles; i++) {
		float rho = 1;
		double depth = H - pos_global[i].z + m_origin.z;
		if (FLUID(info[i])) {
			int fluid_idx = is_inside(m_origin, R, pos_global[i]) ? 0 : 1;
			// hydrostatic density: for the heavy fluid, this is simply computed
			// as the density that gives pressure rho g h, with h depth
			rho = density(depth, fluid_idx);
			// for the bubble, the hydrostatic density must be computed in a slightly
			// more complex way:
			if (fluid_idx == 0) {
				// interface: depth of center of the bubble corrected by
				// R^2 - horizontal offset squared
				// note: no correction by m_origin.z because we are only
				// interested in deltas
				float z_intf = 2*R + sqrtf(R*R
						- (pos_global[i].x)*(pos_global[i].x)
						- (pos_global[i].y)*(pos_global[i].y)
						);
				// pressure at interface, from heavy fluid
				float g = length(physparams()->gravity);
				float P = physparams()->rho0[1]*(H - z_intf)*g;
				// plus hydrostatic pressure from _our_ fluid
				P += physparams()->rho0[0]*(z_intf - pos_global[i].z + m_origin.z)*g;
				rho = density_for_pressure(P, 0);
			}
			info[i]= make_particleinfo(PT_FLUID, fluid_idx, i);
		} else if (BOUNDARY(info[i])) {
			rho = density(depth, 1);
			info[i]= make_particleinfo(PT_BOUNDARY, 1, i);
		}
		pos[i].w *= rho;
		vel[i].w = rho;
	}
}

