#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>

#include "StillWater.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define CENTER_DOMAIN 1
// set to coords (x,y,z) if more accuracy is needed in such point
// (waiting for relative coordinates)
#if CENTER_DOMAIN
#define OFFSET_X (-l/2)
#define OFFSET_Y (-w/2)
#define OFFSET_Z (-h/2)
#else
#define OFFSET_X 0
#define OFFSET_Y 0
#define OFFSET_Z 0
#endif

StillWater::StillWater(GlobalData *_gdata) : XProblem(_gdata)
{
	m_usePlanes = get_option("use-planes", false); // --use-planes true to enable use of planes for boundaries
	const int mlsIters = get_option("mls", 0); // --mls N to enable MLS filter every N iterations
	const int ppH = get_option("ppH", 16); // --ppH N to change deltap to H/N

	// density diffusion terms: 0 none, 1 Molteni & Colagrossi, 2 Ferrari
	const int rhodiff = get_option("density-diffusion", 1);

	SETUP_FRAMEWORK(
		//viscosity<KINEMATICVISC>,
		viscosity<DYNAMICVISC>,
		//viscosity<ARTVISC>,
		boundary<DYN_BOUNDARY>
		//boundary<LJ_BOUNDARY>
	).select_options(
		rhodiff, FlagSwitch<ENABLE_NONE, ENABLE_DENSITY_DIFFUSION, ENABLE_FERRARI>(),
		m_usePlanes, add_flags<ENABLE_PLANES>()
	);

	if (mlsIters > 0)
		addFilter(MLS_FILTER, mlsIters);

	H = 1;

	set_deltap(H/ppH);

	l = w = sqrt(2)*H; h = 1.1*H;

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);
	m_origin = make_double3(OFFSET_X, OFFSET_Y, OFFSET_Z);

	// SPH parameters
	simparams()->dt = 0.00004f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 20;
	simparams()->ferrariLengthScale = H;

	// enlarge the domain to take into account the extra layers of particles
	// of the boundary
	if (simparams()->boundarytype == DYN_BOUNDARY && !m_usePlanes) {
		// number of layers
		dyn_layers = ceil(simparams()->kernelradius*simparams()->sfactor);
		// extra layers are one less (since other boundary types still have
		// one layer)
		double3 extra_offset = make_double3((dyn_layers-1)*m_deltap);
		m_origin -= extra_offset;
		m_size += 2*extra_offset;
	} else {
		dyn_layers = 1;
	}

	simparams()->tend = 100.0;
	if (simparams()->boundarytype == SA_BOUNDARY) {
		simparams()->maxneibsnum = 256; // needed during gamma initialization phase
	};

	// Physical parameters
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	const float g = length(physparams()->gravity);
	const float maxvel = sqrt(2*g*H);
	// purely for cosmetic reason, let's round the soundspeed to the next
	// integer
	const float c0 = ceil(10*maxvel);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, c0);

	physparams()->dcoeff = 5.0f*g*H;

	physparams()->r0 = m_deltap;
	//physparams()->visccoeff = 0.05f;
	set_kinematic_visc(0, 3.0e-2f);
	//set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->epsxsph = 0.5f;

	// Drawing and saving times
	add_writer(VTKWRITER, 1.0);

	// Name of problem used for directory creation
	m_name = "StillWater";

	// Building the geometry
	setPositioning(PP_CORNER);
	// distance between fluid box and wall
	float wd = physparams()->r0;

	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(m_origin), m_size.x, m_size.y, m_size.z);
	disableCollisions(experiment_box);

	m_fluidOrigin = m_origin;
	if (dyn_layers > 1) // shift by the extra offset of the experiment box
		m_fluidOrigin += make_double3((dyn_layers)*m_deltap);
	m_fluidOrigin += make_double3(wd); // one wd space from the boundary
	double shift = 2*wd;
	if (dyn_layers > 1)
		shift = (dyn_layers-1)*m_deltap*2;
	GeometryID fluid = addBox(GT_FLUID, FT_SOLID,
		m_fluidOrigin, l-shift, w-shift, H-shift);

}

void StillWater::copy_planes(PlaneList& planes)
{
	if (!m_usePlanes) return;

	planes.push_back( implicit_plane(0, 0, 1.0, -m_origin.z) );
	planes.push_back( implicit_plane(0, 1.0, 0, -m_origin.x) );
	planes.push_back( implicit_plane(0, -1.0, 0, m_origin.x + w) );
	planes.push_back( implicit_plane(1.0, 0, 0, -m_origin.y) );
	planes.push_back( implicit_plane(-1.0, 0, 0, m_origin.y + l) );
}

