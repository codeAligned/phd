#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <iostream>

#include "DamBreakGate.h"
#include "Cube.h"
#include "Point.h"
#include "Vector.h"
#include "GlobalData.h"
#include "cudasimframework.cu"

#define SIZE_X		(1.60)
#define SIZE_Y		(0.67)
#define SIZE_Z		(0.40)

// default: origin in 0,0,0
#define ORIGIN_X	(0)
#define ORIGIN_Y	(0)
#define ORIGIN_Z	(0)


DamBreakGate::DamBreakGate(GlobalData *_gdata) : XProblem(_gdata)
{
	// Size and origin of the simulation domain
	m_size = make_double3(SIZE_X, SIZE_Y, SIZE_Z + 0.7);
	m_origin = make_double3(ORIGIN_X, ORIGIN_Y, ORIGIN_Z);

	SETUP_FRAMEWORK(
		viscosity<ARTVISC>,//DYNAMICVISC//SPSVISC
		boundary<LJ_BOUNDARY>,
		add_flags<ENABLE_MOVING_BODIES>
	);

	//addFilter(MLS_FILTER, 10);

	// SPH parameters
	set_deltap(0.015f);
	simparams()->dt = 0.0001f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 10;
	simparams()->tend = 10.f;

	// Physical parameters
	H = 0.4f;
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);
	float g = length(physparams()->gravity);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.f);

    //set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;
	physparams()->r0 = m_deltap;

	// BC when using MK boundary condition: Coupled with m_simsparams->boundarytype=MK_BOUNDARY
	#define MK_par 2
	physparams()->MK_K = g*H;
	physparams()->MK_d = 1.1*m_deltap/MK_par;
	physparams()->MK_beta = MK_par;
	#undef MK_par

	set_kinematic_visc(0, 1.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.1);
	add_writer(COMMONWRITER, 0.0);

	// Name of problem used for directory creation
	m_name = "DamBreakGate";

	// Building the geometry
	float r0 = physparams()->r0;
	setPositioning(PP_CORNER);

	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(ORIGIN_X, ORIGIN_Y, ORIGIN_Z), 1.6, 0.67, 0.4);
	disableCollisions(experiment_box);
	GeometryID unfill_top = addBox(GT_FIXED_BOUNDARY, FT_NOFILL,
		Point(ORIGIN_X, ORIGIN_Y, ORIGIN_Z+0.4), 1.6, 0.67, 0.1);
	disableCollisions(unfill_top);
	setEraseOperation(unfill_top, ET_ERASE_BOUNDARY);

	float3 gate_origin = make_float3(0.4 + 2*r0, r0, r0);
	GeometryID gate = addBox(GT_MOVING_BODY, FT_BORDER,
		Point(gate_origin) + Point(ORIGIN_X, ORIGIN_Y, ORIGIN_Z), 0, 0.67-2*r0, 0.4);
	disableCollisions(gate);

	GeometryID obstacle = addBox(GT_FIXED_BOUNDARY, FT_BORDER,
		Point(0.9 + ORIGIN_X, 0.24 + ORIGIN_Y, r0 + ORIGIN_Z), 0.12, 0.12, 0.4 - r0);
	disableCollisions(obstacle);

	GeometryID fluid = addBox(GT_FLUID, FT_SOLID,
		Point(r0 + ORIGIN_X, r0 + ORIGIN_Y, r0 + ORIGIN_Z), 0.4, 0.67 - 2*r0, 0.4 - r0);

	bool wet = false;	// set wet to true have a wet bed experiment
	if (wet) {

		GeometryID fluid1 = addBox(GT_FLUID, FT_SOLID,
			Point(0.4 + 3*r0 + ORIGIN_X, r0 + ORIGIN_Y, r0 + ORIGIN_Z),
			0.5 - 4*r0, 0.67 - 2*r0, 0.03);

		GeometryID fluid2 = addBox(GT_FLUID, FT_SOLID,
			Point(1.02 + r0  + ORIGIN_X, r0 + ORIGIN_Y, r0 + ORIGIN_Z),
			0.58 - 2*r0, 0.67 - 2*r0, 0.03);

		GeometryID fluid3 = addBox(GT_FLUID, FT_SOLID,
			Point(0.9 + ORIGIN_X , m_deltap  + ORIGIN_Y, r0 + ORIGIN_Z),
			0.12, 0.24 - 2*r0, 0.03);

		GeometryID fluid4 = addBox(GT_FLUID, FT_SOLID,
			Point(0.9 + ORIGIN_X , 0.36 + m_deltap  + ORIGIN_Y, r0 + ORIGIN_Z),
			0.12, 0.31 - 2*r0, 0.03);
	}

}

void
DamBreakGate::moving_bodies_callback(const uint index, Object* object, const double t0, const double t1,
			const float3& force, const float3& torque, const KinematicData& initial_kdata,
			KinematicData& kdata, double3& dx, EulerParameters& dr)
{
	const double tstart = 0.1;
	const double tend = 0.4;

	// Computing, at t = t1, new position of center of rotation (here only translation)
	// along with linear velocity
	if (t1 >= tstart && t1 <= tend) {
		kdata.lvel = make_double3(0.0, 0.0, 4.*(t1 - tstart));
		kdata.crot.z = initial_kdata.crot.z + 2.*(t1 - tstart)*(t1 - tstart);
		}
	else
		kdata.lvel = make_double3(0.0f);

	// Computing the displacement of center of rotation between t = t0 and t = t1
	double ti = min(tend, max(tstart, t0));
	double tf = min(tend, max(tstart, t1));
	dx.z = 2.*(tf - tstart)*(tf - tstart) - 2.*(ti - tstart)*(ti - tstart);

	// Setting angular velocity at t = t1 and the rotation between t = t0 and t = 1.
	// Here we have a simple translation movement so the angular velocity is null and
	// the rotation between t0 and t1 equal to identity.
	kdata.avel = make_double3(0.0f);
	dr.Identity();
}


