#include "hip/hip_runtime.h"
/*  Copyright 2011-2013 Alexis Herault, Giuseppe Bilotta, Robert A. Dalrymple, Eugenio Rustico, Ciro Del Negro

    Istituto Nazionale di Geofisica e Vulcanologia
        Sezione di Catania, Catania, Italy

    Università di Catania, Catania, Italy

    Johns Hopkins University, Baltimore, MD

    This file is part of GPUSPH.

    GPUSPH is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    GPUSPH is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with GPUSPH.  If not, see <http://www.gnu.org/licenses/>.
*/


#include <iostream>
#include <stdexcept>

#include "cudasimframework.cu"

#include "Seiche.h"
#include "particledefine.h"
#include "GlobalData.h"

Seiche::Seiche(GlobalData *_gdata) : XProblem(_gdata)
{
	SETUP_FRAMEWORK(
		viscosity<SPSVISC>,
		flags<ENABLE_DTADAPT | ENABLE_PLANES>
	);

	addFilter(MLS_FILTER, 20);

	set_deltap(0.015f);
	H = .5f;
	l = sqrt(2)*H; w = l/2; h = 1.5*H;
	cout << "length= " << l<<"\n";
	cout << "width= " << w <<"\n";
	cout << "h = " << h <<"\n";

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);
	m_origin = make_double3(0.0, 0.0, 0.0);

	// SPH parameters
	simparams()->dt = 0.00004f;
	simparams()->dtadaptfactor = 0.2;
	simparams()->buildneibsfreq = 10;
	simparams()->tend=10.0f;
	simparams()->gcallback=true;

	// Physical parameters
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f); //must be set first
	float g = length(physparams()->gravity);
	add_fluid(1000.0);
	set_equation_of_state(0,  7.0f, 20.f);

    //set p1coeff,p2coeff, epsxsph here if different from 12.,6., 0.5
	physparams()->dcoeff = 5.0f*g*H;
	physparams()->r0 = m_deltap;

	// BC when using MK boundary condition: Coupled with m_simsparams->boundarytype=MK_BOUNDARY
	#define MK_par 2
	physparams()->MK_K = g*H;
	physparams()->MK_d = 1.1*m_deltap/MK_par;
	physparams()->MK_beta = MK_par;
	#undef MK_par

	set_kinematic_visc(0, 5.0e-6f);
	physparams()->artvisccoeff = 0.3f;
	physparams()->smagfactor = 0.12*0.12*m_deltap*m_deltap;
	physparams()->kspsfactor = (2.0/3.0)*0.0066*m_deltap*m_deltap;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;

	// Variable gravity terms:  starting with physparams()->gravity as defined above
	m_gtstart=0.3;
	m_gtend=3.0;

	// Drawing and saving times
	add_writer(VTKWRITER, 0.1);

	// Name of problem used for directory creation
	m_name = "Seiche";

	// Building the geometry
	setPositioning(PP_CORNER);
	// distance between fluid box and wall
	float wd = m_deltap; //Used to be divided by 2

	GeometryID experiment_box = addBox(GT_FIXED_BOUNDARY, FT_BORDER, Point(0, 0, 0), l, w, h);
	disableCollisions(experiment_box);
	GeometryID fluid = addBox(GT_FLUID, FT_SOLID, Point(wd, wd, wd), l-2*wd, w-2*wd, H-2*wd);
}

float3 Seiche::g_callback(const double t)
{
	if(t > m_gtstart && t < m_gtend)
		physparams()->gravity=make_float3(2.*sin(9.8*(t-m_gtstart)), 0.0, -9.81f);
	else
		physparams()->gravity=make_float3(0.,0.,-9.81f);
	return physparams()->gravity;
}

void Seiche::copy_planes(PlaneList& planes)
{
	planes.push_back( implicit_plane(0, 0, 1, 0) );
	planes.push_back( implicit_plane(0, 1, 0, 0) );
	planes.push_back( implicit_plane(0, -1, 0, w) );
	planes.push_back( implicit_plane(1, 0, 0, 0) );
	planes.push_back( implicit_plane(-1, 0, 0, l) );
}
