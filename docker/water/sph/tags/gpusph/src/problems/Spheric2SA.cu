#include "hip/hip_runtime.h"
#include <string>
#include <iostream>

#include "Spheric2SA.h"
#include "GlobalData.h"
#include "cudasimframework.cu"
#include "textures.cuh"
#include "utils.h"

#define USE_PLANES 0

Spheric2SA::Spheric2SA(GlobalData *_gdata) : XProblem(_gdata)
{
	SETUP_FRAMEWORK(
		viscosity<DYNAMICVISC>,
		boundary<SA_BOUNDARY>,
		periodicity<PERIODIC_NONE>,
		kernel<WENDLAND>,
		add_flags<ENABLE_FERRARI | ENABLE_GAMMA_QUADRATURE>
	);

	set_deltap(0.02715f);

	size_t water = add_fluid(1000.0);
	set_equation_of_state(water,  7.0f, 130.f);
	set_kinematic_visc(water, 1.0e-2f);
	physparams()->gravity = make_float3(0.0, 0.0, -9.81f);

	simparams()->tend = 5.0;
	addPostProcess(SURFACE_DETECTION);
	addPostProcess(TESTPOINTS);
	H = 0.55;
	l = 3.5+0.02; w = 1.0+0.02; h = 2.0;
	m_origin = make_double3(-0.01, -0.01, -0.01);
	simparams()->ferrariLengthScale = 0.161f;
	simparams()->maxneibsnum = 240;

	// SPH parameters
	simparams()->dt = 0.00004f;
	simparams()->dtadaptfactor = 0.3;
	simparams()->buildneibsfreq = 1;
	simparams()->nlexpansionfactor = 1.1;

	// Size and origin of the simulation domain
	m_size = make_double3(l, w ,h);

	// Physical parameters
	float g = length(physparams()->gravity);

	physparams()->dcoeff = 5.0f*g*H;

	physparams()->r0 = m_deltap;

	physparams()->artvisccoeff = 0.3f;
	physparams()->epsartvisc = 0.01*simparams()->slength*simparams()->slength;
	physparams()->epsxsph = 0.5f;

	// Drawing and saving times
	add_writer(VTKWRITER, 1e-2f);

	// Name of problem used for directory creation
	m_name = "Spheric2SA";

	// Building the geometry
	addHDF5File(GT_FLUID, Point(0,0,0), "./data_files/Spheric2/0.spheric2.fluid.h5sph", NULL);

	GeometryID container =
		addHDF5File(GT_FIXED_BOUNDARY, Point(0,0,0), "./data_files/Spheric2/0.spheric2.boundary.kent0.h5sph", NULL);
	disableCollisions(container);

	// Add water level gages
	add_gage(m_origin + make_double3(2.724, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(2.228, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(1.732, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));
	add_gage(m_origin + make_double3(0.582, 0.5, 0.0) + make_double3(0.01, 0.01, 0.01));

	// Pressure probes
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.021) + make_double3(0.01, 0.01, 0.01)); // the (0.01,0.01,0.01) vector accounts for the slightly shifted origin
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.061) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.101) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.3955, 0.5, 0.141) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4165, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4565, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.4965, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));
	addTestPoint(m_origin + make_double3(2.5365, 0.5, 0.161) + make_double3(0.01, 0.01, 0.01));

}

void
Spheric2SA::initializeParticles(BufferList &buffers, const uint numParticles)
{
	printf("k and epsilon initialization...\n");

	float4 *vel = buffers.getData<BUFFER_VEL>();
	particleinfo *info = buffers.getData<BUFFER_INFO>();
	double4 *pos = buffers.getData<BUFFER_POS_GLOBAL>();
	float *k = buffers.getData<BUFFER_TKE>();
	float *epsilon = buffers.getData<BUFFER_EPSILON>();

	for (uint i = 0; i < numParticles; i++) {
		const float Ti = 0.01f;
		const float u = 1.0f; // TODO set according to initial velocity
		const float L = 1.0f; // TODO set according to geometry
		if (k && epsilon) {
			k[i] = fmaxf(1e-5f, 3.0f/2.0f*(u*Ti)*(u*Ti));
			epsilon[i] = fmaxf(1e-5f, 2.874944542f*k[i]*u*Ti/L);
			//k[i] = k0;
			//e[i] = 1.0f/0.41f/fmax(1.0f-fabs(z),0.5f*(float)m_deltap);
		}
	}
}

uint
Spheric2SA::max_parts(uint numpart)
{
	// gives an estimate for the maximum number of particles
	return numpart;
}

void Spheric2SA::fillDeviceMap()
{
	fillDeviceMapByAxis(X_AXIS);
}
